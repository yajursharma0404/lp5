
#include <hip/hip_runtime.h>
#include <iostream>
void addVectors(int *A, int *B, int *C, int n)
{
    for (int i = 0; i < n; i++)
    {
        C[i] = A[i] + B[i];
    }
}
int main()
{
    int n = 1000000;
    int *A, *B, *C;
    A = new int[n];
    B = new int[n];
    C = new int[n];
    for (int i = 0; i < n; i++)
    {
        A[i] = i;
        B[i] = i * 2;
    }
    addVectors(A, B, C, n);
    for (int i = 0; i < 10; i++)
    {
        std::cout << C[i] << " ";
    }
    std::cout << std::endl;
    delete[] A;
    delete[] B;
    delete[] C;
    return 0;
}

/*
without gpu
1. sudo apt install nvidia-cuda-toolkit
2. wget https://developer.download.nvidia.com/compute/cuda/12.4.1/local_installers/cuda_12.4.1_550.54.15_linux.run
3. nvcc -arch=sm_60 addVector.cu -o addVector
4. ./addVector
*/