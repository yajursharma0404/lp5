
#include <hip/hip_runtime.h>
#include <iostream>
void matmul(int *A, int *B, int *C, int N)
{
    for (int Row = 0; Row < N; Row++)
    {
        for (int Col = 0; Col < N; Col++)
        {
            int Pvalue = 0;
            for (int k = 0; k < N; k++)
            {
                Pvalue += A[Row * N + k] * B[k * N + Col];
            }
            C[Row * N + Col] = Pvalue;
        }
    }
}
int main()
{
    int N = 512;
    int size = N * N * sizeof(int);
    int *A, *B, *C;
    A = new int[size];
    B = new int[size];
    C = new int[size];
    // Initialize matrices A and B
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            A[i * N + j] = i * N + j;
            B[i * N + j] = j * N + i;
        }
    }
    // Perform matrix multiplication
    matmul(A, B, C, N);
    // Print the result (just printing the first 10x10 elements)
    for (int i = 0; i < 10; i++)
    {
        for (int j = 0; j < 10; j++)
        {
            std::cout << C[i * N + j] << " ";
        }
        std::cout << std::endl;
    }
    // Free memory
    delete[] A;
    delete[] B;
    delete[] C;
    return 0;
}

/*
without gpu
1. sudo apt install nvidia-cuda-toolkit
2. wget https://developer.download.nvidia.com/compute/cuda/12.4.1/local_installers/cuda_12.4.1_550.54.15_linux.run
3. nvcc -arch=sm_60 addVector.cu -o addVector
4. ./addVector
*/